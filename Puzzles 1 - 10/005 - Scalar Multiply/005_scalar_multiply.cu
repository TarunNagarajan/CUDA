#include <iostream>
#include <hip/hip_runtime.h>

#define N 10
#define THREADS_PER_BLOCK 256

// Problem: | 5 | Scalar Multiply | Multiply each element of a float array by a scalar value on the GPU.    

__global__ void scalarMultiply(const float* input, float *output, const float scalar, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx] = input[idx] * scalar;
    }
}

int main() {
    // allocate the host arrays, memory. (on the cpu)
    int bytes = N * sizeof(float);
    float scalar = 2.0f;

    float *h_input = new float[N];
    float *h_output = new float[N];

    // allocate the device arrays, memory. (gpu)
    float *d_input;
    float *d_output;

    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, bytes);

    for (int i = 0; i < N; ++i) {
        h_input[i] = static_cast<float>(i + 1); // [1, 2, 3, 4, ...]
    }

    // input to device.
    hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

    // launch kernel.
    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK; 
    scalarMultiply<<<blocks, THREADS_PER_BLOCK>>>(d_input, d_output, scalar, N);
    hipDeviceSynchronize();

    // result to host.
    hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost);

    // print the result.
    std::cout << "Input: ";
    for (int i = 0; i < N; i++) {
        std::cout << h_input[i] << " ";
    }

    std::cout << "\nOutput: ";
    for (int i = 0; i < N; i++) {
        std::cout << h_output[i] << " ";
    }

    delete[] h_input;
    delete[] h_output;

    hipFree(d_input);
    hipFree(d_output);

    return 0; 

}
