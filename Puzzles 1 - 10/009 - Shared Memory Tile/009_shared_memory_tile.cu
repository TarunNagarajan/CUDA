#include <iostream>
#include <hip/hip_runtime.h>

#define TILE_SIZE 8

__global__ void sharedTile(float* out, const float* in) {
    __shared__ float tile[TILE_SIZE];
    int idx = threadIdx.x;

    if (idx < TILE_SIZE) {
        tile[idx] = in[idx];
        __syncthreads();

        tile[idx] *= tile[idx]; // transform: squaring each element of the array. 
        __syncthreads();

        out[idx] = tile[idx];
    }  
}

int main() {
    // we're allocating memory on-device for this one. 
    float h_in[TILE_SIZE] = {1, 2, 3, 4, 5, 6, 7, 8}; // 8 TILE_SIZE
    float h_out[TILE_SIZE];

    float* d_in;
    float* d_out;

    // number of bytes
    size_t bytes = sizeof(float) * TILE_SIZE;

    // on-device memory allocation
    hipMalloc((void**)&d_in, bytes);
    hipMalloc((void**)&d_out, bytes);

    hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice);

    sharedTile<<<1, TILE_SIZE>>>(d_out, d_in);
    hipDeviceSynchronize();

    hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost);

    std::cout << "Transformed output:\n";
    for (int i = 0; i < TILE_SIZE; ++i) {
        std::cout << h_out[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_in);
    hipFree(d_out);

    return 0; 


}
