#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cfloat>

__global__ void smem_layernorm(float* matrix, float* output, int m, int n, float epsilon) {
    int row = blockIdx.x;
    int col = threadIdx.x;
    int idx = (row * n) + col;

    extern __shared__ float smem[];

    float val = matrix[idx];
    smem[col] = val;
    __syncthreads();

    // STRIDE I: MEAN
    for (stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (col < stride) {
            smem[col] += smem[col + stride];
        }
        __syncthreads();
    }

    // thread 0 is allowed to calculate the mean and store it in smem[0] for broadcasting
    if (col == 0) {
        smem[0] = smem[0] / n;
    }
    __syncthreads();

    float mean = smem[0];

    // STRIDE II: VARIANCE
    float diff = val - mean;
    smem[col] = diff * diff;
    __syncthreads(); 

    for (stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (col < stride) {
            smem[col] += smem[col + stride];
        }
        __syncthreads();
    }

    // thread 0 is allowed to calculate the mean (in this case, the sum of squared differences divided by n(samples)) and store it in smem[0]
    // for broadcasting
    if (col == 0) {
        smem[0] = smem[0] / n;
    }
    __syncthreads();

    float variance = smem[0];

    float stdev = sqrtf(variance + epsilon);
    output[idx] = (val - mean) / stdev;
}

#define CHECK_CUDA(call)                                                    \
    do {                                                                    \
        hipError_t err = (call);                                           \
        if (err != hipSuccess) {                                           \
            std::cerr << "CUDA error: " << hipGetErrorString(err)          \
                      << " at " << __FILE__ << ":" << __LINE__ << std::endl;\
            std::exit(EXIT_FAILURE);                                        \
        }                                                                   \
    } while (0)

int main() {
    int m = 1024;
    int n = 1024;
    size_t SIZE = m * n;

    hipEvent_t startH2D, startKernel, startD2H, stop;
    CHECK_CUDA(hipEventCreate(&startH2D)); 
    CHECK_CUDA(hipEventCreate(&startKernel));
    CHECK_CUDA(hipEventCreate(&startD2H));
    CHECK_CUDA(hipEventCreate(&stop));

    dim3 threadsPerBlock(n);
    dim3 blocksPerGrid(m);

    size_t SMEM_SIZE = threadsPerBlock.x * sizeof(float);

    float* h_matrix = new float[SIZE];
    for (int i = 0; i < SIZE; i++) {
        h_matrix[i] = static_cast<float>(i % 100); // fix: i -> i % 100 for better test values
    }

    float* d_matrix;
    CHECK_CUDA(hipMalloc((void**)&d_matrix, SIZE * sizeof(float)));

    float* d_output;
    CHECK_CUDA(hipMalloc((void**)&d_output, SIZE * sizeof(float)));

    CHECK_CUDA(hipEventRecord(startH2D));
    CHECK_CUDA(hipMemcpy(d_matrix, h_matrix, SIZE * sizeof(float), hipMemcpyHostToDevice));

    CHECK_CUDA(hipEventRecord(startKernel));
    smem_layernorm<<<blocksPerGrid, threadsPerBlock>>>(d_matrix, d_output, m, n, 1e-6f);
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipEventRecord(startD2H));
    float* h_output = new float[SIZE];
    CHECK_CUDA(hipMemcpy(h_output, d_output, SIZE * sizeof(float), hipMemcpyDeviceToHost));

    CHECK_CUDA(hipEventRecord(stop)); 
    CHECK_CUDA(hipEventSynchronize(stop));

    std::cout << "First 10 normalized values from the first row:\n";
    for (int i = 0; i < 10; i++) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    float h2dTime, kernelTime, d2hTime; 
    CHECK_CUDA(hipEventElapsedTime(&h2dTime, startH2D, startKernel));
    CHECK_CUDA(hipEventElapsedTime(&kernelTime, startKernel, startD2H));
    CHECK_CUDA(hipEventElapsedTime(&d2hTime, startD2H, stop)); 

    std::cout << "\n";
    std::cout << "NVIDIA GTX TITAN X\n";
    std::cout << "Host to Device Transfer: " << h2dTime << " ms" << std::endl;
    std::cout << "Kernel Execution: " << kernelTime << " ms" << std::endl;
    std::cout << "Device to Host Transfer: " << d2hTime << " ms" << std::endl;

    delete[] h_matrix;
    delete[] h_output;
    CHECK_CUDA(hipFree(d_matrix));
    CHECK_CUDA(hipFree(d_output));

    return 0;
}
