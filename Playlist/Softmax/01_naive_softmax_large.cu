#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <ctime>
#include <iomanip>
#include <cfloat>

__global__ void naive_softmax(const float* A, float* B, int M, int N) {
    int row = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < M) {
        float x_max = -FLT_MAX;

        // Step 1: Find max value in row
        for (int col = 0; col < N; col++) {
            int i = row * N + col;
            x_max = fmaxf(x_max, A[i]);
        }

        // Step 2: Compute denominator using double for better precision
        double norm = 0.0;
        for (int col = 0; col < N; col++) {
            int i = row * N + col;
            norm += exp((double)(A[i] - x_max));
        }

        // Step 3: Compute softmax output
        for (int col = 0; col < N; col++) {
            int i = row * N + col;
            B[i] = (float)(exp((double)(A[i] - x_max)) / norm);
        }
    }
}

void CHECK_CUDA(hipError_t call) {
    if (call != hipSuccess) {
        std::cerr << "Error at " << __LINE__ << ": " << hipGetErrorString(hipGetLastError()) << std::endl;
        exit(1);
    }
}

int main() {
    constexpr int M = 1024;
    constexpr int N = 32768;

    const int SIZE = M * N * sizeof(float);
    float* h_input = new float[M * N];
    float* h_output = new float[M * N];

    // Safer random values: avoid extreme ranges
    srand(42);
    for (int i = 0; i < M * N; i++) {
        h_input[i] = ((float)rand() / RAND_MAX) * 10.0f - 5.0f; // range [-5, 5]
    }

    float* d_A;
    float* d_B;

    hipEvent_t startH2D, startKernel, startD2H, stop;
    hipEventCreate(&startH2D);
    hipEventCreate(&startKernel);
    hipEventCreate(&startD2H);
    hipEventCreate(&stop);

    CHECK_CUDA(hipMalloc(&d_A, SIZE));
    CHECK_CUDA(hipMalloc(&d_B, SIZE));

    int BLOCK_SIZE = 128;
    int BLOCKS = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // time window: 1 (copy input from host to device)
    hipEventRecord(startH2D);
    CHECK_CUDA(hipMemcpy(d_A, h_input, SIZE, hipMemcpyHostToDevice));
    hipEventRecord(startKernel);

    // time window: 2 (launch kernel)
    naive_softmax<<<BLOCKS, BLOCK_SIZE>>>(d_A, d_B, M, N);
    CHECK_CUDA(hipDeviceSynchronize());
    hipEventRecord(startD2H);

    // time window: 3 (copy output back to host)
    CHECK_CUDA(hipMemcpy(h_output, d_B, SIZE, hipMemcpyDeviceToHost));
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float h2dTime, kernelTime, d2hTime;
    hipEventElapsedTime(&h2dTime, startH2D, startKernel);
    hipEventElapsedTime(&kernelTime, startKernel, startD2H);
    hipEventElapsedTime(&d2hTime, startD2H, stop);

    std::cout << "Softmax Output:\n";
    for (int i = 0; i < 4; i++) {
        std::cout << "Row " << std::setw(2) << i << ": ";
        for (int j = 0; j < 5; j++) {
            float val = h_output[i * N + j];
            std::cout << std::fixed << std::setprecision(4) << std::setw(8) << val << " ";
        }
        std::cout << "\n";
    }

    // Validate softmax sum per row
    for (int i = 0; i < 4; i++) {
        double row_sum = 0.0;
        for (int j = 0; j < N; j++) {
            row_sum += h_output[i * N + j];
        }
        std::cout << "Row " << i << " sum: " << row_sum << "\n";
    }

    std::cout << "\n";
    std::cout << "Host to Device Transfer: " << h2dTime << " ms" << std::endl;
    std::cout << "Kernel Execution: " << kernelTime << " ms" << std::endl;
    std::cout << "Device to Host Transfer: " << d2hTime << " ms" << std::endl;

    delete[] h_input;
    delete[] h_output;
    hipFree(d_A);
    hipFree(d_B);
    hipEventDestroy(startH2D);
    hipEventDestroy(startKernel);
    hipEventDestroy(startD2H);
    hipEventDestroy(stop);

    return 0;
}

/*

Running in FUNCTIONAL mode...
Compiling...
Executing...
Host to Device Transfer: 49.0650 ms
Kernel Execution: 1408.4968 ms
Device to Host Transfer: 108.0687 ms
Exit status: 0

*/
